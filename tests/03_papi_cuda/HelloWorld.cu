#include "hip/hip_runtime.h"
/****************************/
/* THIS IS OPEN SOURCE CODE */
/****************************/

/** 
 * @file    HelloWorld.c
 * CVS:     $Id$
 * @author  Heike Jagode
 *          jagode@eecs.utk.edu
 * Mods:	<your name here>
 *			<your email address>
 * test case for Example component 
 * 
 *
 * @brief
 *  This file is a very simple HelloWorld C example which serves (together
 *	with its Makefile) as a guideline on how to add tests to components.
 *  The papi configure and papi Makefile will take care of the compilation
 *	of the component tests (if all tests are added to a directory named
 *	'tests' in the specific component dir).
 *	See components/README for more details.
 *
 *	The string "Hello World!" is mangled and then restored.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <papi.h>

#define NUM_EVENTS 1
#define PAPI

// Prototypes
__global__ void helloWorld(char*);


// Host function
int main(int argc, char** argv)
{
#ifdef PAPI
	int retval, i;
	int EventSet = PAPI_NULL;
	long long values[NUM_EVENTS];
	/* REPLACE THE EVENT NAME 'PAPI_FP_OPS' WITH A CUDA EVENT 
	   FOR THE CUDA DEVICE YOU ARE RUNNING ON.
	   RUN papi_native_avail to get a list of CUDA events that are 
	   supported on your machine */
    char *EventName[] = { "PAPI_FP_OPS" };
	int events[NUM_EVENTS];
	
	/* PAPI Initialization */
	retval = PAPI_library_init( PAPI_VER_CURRENT );
	if( retval != PAPI_VER_CURRENT )
		fprintf( stderr, "PAPI_library_init failed\n" );
	
	printf( "PAPI_VERSION     : %4d %6d %7d\n",
			PAPI_VERSION_MAJOR( PAPI_VERSION ),
			PAPI_VERSION_MINOR( PAPI_VERSION ),
			PAPI_VERSION_REVISION( PAPI_VERSION ) );
	
	/* convert PAPI native events to PAPI code */
	for( i = 0; i < NUM_EVENTS; i++ ){
		retval = PAPI_event_name_to_code( EventName[i], &events[i] );
		if( retval != PAPI_OK )
			fprintf( stderr, "PAPI_event_name_to_code failed\n" );
		else
			printf( "Name %s --- Code: %x\n", EventName[i], events[i] );
	}

	retval = PAPI_create_eventset( &EventSet );
	if( retval != PAPI_OK )
		fprintf( stderr, "PAPI_create_eventset failed\n" );
	
	retval = PAPI_add_events( EventSet, events, NUM_EVENTS );
	if( retval != PAPI_OK )
		fprintf( stderr, "PAPI_add_events failed\n" );
	
	retval = PAPI_start( EventSet );
	if( retval != PAPI_OK )
		fprintf( stderr, "PAPI_start failed\n" );
#endif


	#define N 10

	int j;
	
	// desired output
	int str[] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};

	// mangle contents of output
	// the null character is left intact for simplicity
	for(j = 0; j < 12; j++) {
		str[j] -= j;
		//printf("str=%s\n", str);
	}

	if (argc != 4) {
		printf("Usage: ./a.out <threads> <blocks> <counter_name>")
		return -2;
	}

	int threads = atoi(argv[1]);
	int blocks = atoi(argv[2]);
	
	// allocate memory on the device
	char *d_str;

	hipMalloc((void**)&d_str, sizeof(int) * N);
	
	// copy the string to the device
	hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);
	

	// invoke the kernel
	kernel<<< threads, blocks >>>(d_str);

	// retrieve the results from the device
	hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);
	
	// free up the allocated memory on the device
	hipFree(d_str);
	
	printf("END: %s\n", str);

	
#ifdef PAPI
	retval = PAPI_stop( EventSet, values );
	if( retval != PAPI_OK )
		fprintf( stderr, "PAPI_stop failed\n" );

	for( i = 0; i < NUM_EVENTS; i++ )
		printf( "%12lld \t\t --> %s \n", values[i], EventName[i] );
#endif

	return 0;
}


// Device kernel
__global__ void kernel(int *arr) {
	// should return 11
	__prof_trigger(00);

	int id = threadIdx.x + blockDim.x * blockIdx.x;
	if (id >= N) return;

	// should yield 10
	__prof_trigger(01);

	if (arr[id] < 4) __prof_trigger(02); //should yield 4
	else             __prof_trigger(03); //should yield 6
}