#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cupti.h>
#include <stdio.h>

#define N 10
#define THREADS 11

#define EVENT_NAME_TESLA "instructions"
#define EVENT_NAME_FERMI "inst_executed"

#define CHECK_CU_ERROR(err, cufunc)										\
	if (err != hipSuccess) { 											\
		printf("%s:%d: error %d for CUDA Driver API function '%s'\n",	\
				__FILE__, __LINE__, err, cufunc);						\
		exit(-1);														\
	}

#define CHECK_CUPTI_ERROR(err, cuptifunc)								\
	if (err != CUPTI_SUCCESS) {											\
		const char *errstr;												\
		cuptiGetResultString(err, &errstr);								\
		printf("%s:%d:Error %s for CUPTI API function '%s'\n",			\
				__FILE__, __LINE__, errstr, cuptifunc);					\
		exit(-1);														\
	}

typedef struct cupti_eventData_st {
	CUpti_EventGroup eventGroup;
	CUpti_EventID eventId;
} cupti_eventData;

// Structure to hold data collected by callback
typedef struct RuntimeApiTrace_st {
	cupti_eventData *eventData;
	uint64_t eventVal;
} RuntimeApiTrace_t;

void CUPTIAPI getEventValueCallback(
						void *userdata,
						CUpti_CallbackDomain domain,
						CUpti_CallbackId cbid,
						const CUpti_CallbackData *cbInfo) {

	CUptiResult cuptiErr;
	RuntimeApiTrace_t *traceData = (RuntimeApiTrace_t*) userdata;
	size_t bytesRead;

	// This callback is enabled for launch so we shouldn't see anything else.
	if (cbid != CUPTI_RUNTIME_TRACE_CBID_cudaLaunch_v3020) {
		printf("%s:%d: unexpected cbid %d\n", __FILE__, __LINE__, cbid);
		exit(-1);
	}

	switch(cbInfo->callbackSite) {
		case CUPTI_API_ENTER:
			hipDeviceSynchronize();
			cuptiErr = cuptiSetEventCollectionMode(cbInfo->context, CUPTI_EVENT_COLLECTION_MODE_KERNEL);
			CHECK_CUPTI_ERROR(cuptiErr, "cuptiSetEventCollectionMode");
			cuptiErr = cuptiEventGroupEnable(traceData->eventData->eventGroup);
			CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupEnable");
			break;

		case CUPTI_API_EXIT:
			bytesRead = sizeof(uint64_t);
			hipDeviceSynchronize();
			cuptiErr = cuptiEventGroupReadEvent(traceData->eventData->eventGroup, CUPTI_EVENT_READ_FLAG_NONE, traceData->eventData->eventId, &bytesRead, &traceData->eventVal);
			CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupReadEvent");
			cuptiErr = cuptiEventGroupDisable(traceData->eventData->eventGroup);
			CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupDisable");
			break;
	}
}

static void displayEventVal(RuntimeApiTrace_t *trace, char *eventName) {
	printf("Event Name: %s \n", eventName);
	printf("Event Value: %llu\n", (unsigned long long) trace->eventVal);
}

__global__ void kernel(int *arr) {

	int id = threadIdx.x + blockDim.x * blockIdx.x;
	if (id >= N) return;

	arr[id] = 1;
}

int main() {
	int deviceCount;
	hipCtx_t context = 0;
	hipDevice_t dev = 0;
	char deviceName[32];
	char *eventName;
	CUptiResult cuptiErr;
	CUpti_SubscriberHandle subscriber;
	cupti_eventData cuptiEvent;
	RuntimeApiTrace_t trace;
	int cap_major, cap_minor;

	hipError_t err = hipInit(0);
	CHECK_CU_ERROR(err, "hipInit");

	err = hipGetDeviceCount(&deviceCount);
	CHECK_CU_ERROR(err, "hipGetDeviceCount");

	if (deviceCount == 0) {
		printf("There is no device supporting CUDA.\n");
		return -2;
	}

	err = hipDeviceGet(&dev, 0);
	CHECK_CU_ERROR(err, "hipDeviceGet");

	err = hipDeviceGetName(deviceName, 32, dev);
	CHECK_CU_ERROR(err, "hipDeviceGetName");

	err = hipDeviceComputeCapability(&cap_major, &cap_minor, dev);
	CHECK_CU_ERROR(err, "hipDeviceComputeCapability");

	printf("CUDA Device Name: %s\n", deviceName);
	printf("CUDA Capability: %d.%d\n", cap_major, cap_minor);

	err = hipCtxCreate(&context, 0, dev);
	CHECK_CU_ERROR(err, "hipCtxCreate");

	cuptiErr = cuptiEventGroupCreate(context, &cuptiEvent.eventGroup, 0);
	CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupCreate");

	if (cap_major > 1)
		eventName = EVENT_NAME_FERMI;
	else
		eventName = EVENT_NAME_TESLA;

	cuptiErr = cuptiEventGetIdFromName(dev, eventName, &cuptiEvent.eventId);
	if (cuptiErr != CUPTI_SUCCESS) {
		printf("Invalid eventName: %s\n", eventName);
		return -1;
	}

	cuptiErr = cuptiEventGroupAddEvent(cuptiEvent.eventGroup, cuptiEvent.eventId);
	CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupAddEvent");

	trace.eventData = &cuptiEvent;

	cuptiErr = cuptiSubscribe(&subscriber, (CUpti_CallbackFunc)getEventValueCallback, &trace);
	CHECK_CUPTI_ERROR(cuptiErr, "cuptiSubscribe");

	cuptiErr = cuptiEnableCallback(1, subscriber, CUPTI_CB_DOMAIN_RUNTIME_API, CUPTI_RUNTIME_TRACE_CBID_cudaLaunch_v3020);
	CHECK_CUPTI_ERROR(cuptiErr, "cuptiEnableCallback");


	int host_arr[] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
	int *dev_arr;
	

	hipMalloc(&dev_arr, sizeof(int) * N);
	hipMemcpy(dev_arr, &host_arr, sizeof(int) * N, hipMemcpyHostToDevice);
	kernel<<< THREADS, 1 >>>(dev_arr);

	displayEventVal(&trace, eventName);
	trace.eventData = NULL;

	cuptiErr = cuptiEventGroupRemoveEvent(cuptiEvent.eventGroup, cuptiEvent.eventId);
	CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupRemoveEvent");

	cuptiErr = cuptiEventGroupDestroy(cuptiEvent.eventGroup);
	CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupDestroy");

	cuptiErr = cuptiUnsubscribe(subscriber);
	CHECK_CUPTI_ERROR(cuptiErr, "cuptiUnsubscribe");

	hipDeviceSynchronize();
}
